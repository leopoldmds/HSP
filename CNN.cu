
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>


__global__ void cuda_hello(){
    printf("Hello World \n");
}

void MatrixInit(float *M, int n, int p){
    for (int i=0; i<n*p+1;i++){
            M[i]=(float)rand()/(float)(RAND_MAX)*2-1;
        }
}

void MatrixPrint(float *M, int n, int p){
    printf(" %f ",M[0]);
    for (int i=2;i<n*p+1;i++){
    if(i%p!=0){
    printf(" %f ",M[i]);
    }
    else{
    printf(" %f \n",M[i]);}
    }
    printf("\n"); 
}

void MatrixAdd(float *M1, float *M2, float *Mout, int n, int p){
    for (int i=0; i<n*p+1;i++){
        Mout[i]=M1[i]+M2[i];
    }
}



int main(){
    int n = 4;
    int p = 4;
    float *M1 = (float *)malloc(n*p*sizeof(float));
    float *M2 = (float *)malloc(n*p*sizeof(float));
    float *Mout = (float *)malloc(n*p*sizeof(float));
    MatrixInit(M1,n,p);
    MatrixInit(M2,n,p);
    MatrixAdd(M1,M2,Mout,n,p);
    MatrixPrint(Mout,n,p);
    hipDeviceSynchronize();
    return 0;
}
